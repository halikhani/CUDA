#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vecAdd(const float* A, const float* B, float* C, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N){
        C[i] = A[i] + B[i];
    }
}

int main(){
    const int N = 10;
    float A[N], B[N], C[N];
    float *d_a, *d_b, *d_c;

    // Initialize inputs
    for (int i = 0; i < N; i++) {
        A[i] = i * 1.0f;
        B[i] = i * 2.0f;
    }

    hipMalloc(&d_a, N*sizeof(float));
    hipMalloc(&d_b, N*sizeof(float));
    hipMalloc(&d_c, N*sizeof(float));

    hipMemcpy(d_a, A, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, N*sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    // int gridsize = ceil(N/blocksize);
    int gridSize = (N + blockSize - 1) / blockSize;

    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);
    // Wait for GPU to finish
    hipDeviceSynchronize();

    hipMemcpy(C, d_c, N*sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    std::cout << "A + B = C:\n";
    for (int i = 0; i < N; i++) {
        std::cout << A[i] << " + " << B[i] << " = " << C[i] << std::endl;
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}